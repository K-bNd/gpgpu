#include "hip/hip_runtime.h"
#include "filter_impl.h"

#include <cassert>
#include <chrono>
#include <thread>
#include <cstdio>
#include "logo.h"

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char *const func, const char *const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::fprintf(stderr, "CUDA Runtime Error at: %s: %d\n", file, line);
        std::fprintf(stderr, "%s %s\n", hipGetErrorString(err), func);
        // We don't exit when we encounter CUDA errors in this example.
        std::exit(EXIT_FAILURE);
    }
}

struct rgb
{
    uint8_t r, g, b;
};

struct lab
{
    uint8_t L, a, b;
};

__constant__ uint8_t *logo;

/// @brief Black out the red channel from the video and add EPITA's logo
/// @param buffer
/// @param width
/// @param height
/// @param stride
/// @param pixel_stride
/// @return
__global__ void remove_red_channel_inp(std::byte *buffer, int width, int height, int stride)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= width || y >= height)
        return;

    rgb *lineptr = (rgb *)(buffer + y * stride);
    if (y < logo_height && x < logo_width)
    {
        float alpha = logo[y * logo_width + x] / 255.f;
        lineptr[x].r = 0;
        lineptr[x].g = uint8_t(alpha * lineptr[x].g + (1 - alpha) * 255);
        lineptr[x].b = uint8_t(alpha * lineptr[x].b + (1 - alpha) * 255);
    }
    else
    {
        lineptr[x].r = 0;
    }
}

__global__ void erosion(std::byte *buffer, int width, int height, int stride)
{
    int rayon = 3;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= width || y >= height)
        return;
    rgb *lineptr = (rgb *)(buffer + y * stride);

    for (int dy = -rayon; dy <= rayon; dy++)
    {
        rgb *lineptr_comp = (rgb *)(buffer + (y + dy) * stride);
        for (int dx = -rayon; dx <= rayon; dx++)
        {
            if (y + dy < 0 || y + dy >= height || x + dx < 0 || x + dx >= width)
                continue;
            uint8_t sum = lineptr_comp[x + dx].r + lineptr_comp[x + dx].g + lineptr_comp[x + dx].b;
            if (sum < lineptr[x].r + lineptr[x].g + lineptr[x].b)
                lineptr[x] = lineptr_comp[x + dx];
        }
    }
}

__device__ lab rgbToLab(rgb color)
{
    // Convert RGB to XYZ
    uint8_t X = (0.4124564 * color.r + 0.3575761 * color.g + 0.1804674 * color.b) / 255.0;
    uint8_t Y = (0.2126729 * color.r + 0.7152282 * color.g + 0.072099 * color.b) / 255.0;
    uint8_t Z = (0.0193339 * color.r + 0.1191920 * color.g + 0.9503041 * color.b) / 255.0;

    // Convert XYZ to CIE L*a*b*
    uint8_t L = 116.0 * pow(Y / 0.008856, 1.0 / 3.0) - 16.0;
    uint8_t a = 500.0 * (pow(X / 0.950456, 1.0 / 3.0) - pow(Y / 1.0, 1.0 / 3.0));
    uint8_t b = 200.0 * (pow(Y / 1.0, 1.0 / 3.0) - pow(Z / 1.07700, 1.0 / 3.0));

    return {.L = L, .a = a, .b = b};
}

__device__ rgb labToRgb(lab &pixel)
{
    // Convert CIE L*a*b* to XYZ
    double X = (pixel.L + 16.0) / 116.0;
    double Y = (X * 0.008856 + 16.0) / 116.0;
    double Z = Y / 1.181678;

    double r = 3.240479 * X - 1.537383 * Y - 0.498531 * Z;
    double g = -0.969256 * X + 1.875991 * Y + 0.041556 * Z;
    double b = 0.055648 * X - 0.201966 * Y + 1.253272 * Z;

    // Convert XYZ to RGB
    r = 255.0 * r;
    g = 255.0 * g;
    b = 255.0 * b;

    // Clamp values to valid RGB range
    r = max(0.0, min(255.0, r));
    g = max(0.0, min(255.0, g));
    b = max(0.0, min(255.0, b));

    return {.r = (uint8_t)r, .g = (uint8_t)g, .b = (uint8_t)b};
}

__device__ rgb computeDistance(rgb a, rgb b)
{
    // Implement your RGB to Lab conversion here
    // This is a simplified version, you may want to use a more accurate conversion method
    lab lab_a = rgbToLab(a);

    lab lab_b = rgbToLab(b);

    uint8_t delta = sqrt(pow(lab_b.L - lab_a.L, 2.0) + pow(lab_b.a - lab_a.a, 2.0) + pow(lab_b.b - lab_a.b, 2.0));

    lab result = {delta, delta, delta};
    return labToRgb(result);
}

// CUDA kernel to perform image difference
__global__ void imageDiff(std::byte *buffer, std::byte *background, int width, int height, int stride)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= width || y >= height)
        return;

    rgb *lineptr = (rgb *)(buffer + y * stride);
    rgb *lineptr_background = (rgb *)(background + y * stride);

    lineptr[x] = computeDistance(lineptr[x], lineptr_background[x]);
}

// CUDA kernel to update the background
__global__ void updateBackground(std::byte *buffer, std::byte *background, int width, int height, int stride)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= width || y >= height)
        return;

    rgb *lineptr = (rgb *)(buffer + y * stride);
    rgb *lineptr_background = (rgb *)(background + y * stride);

    lineptr_background[x] = {
        static_cast<uint8_t>((lineptr_background[x].r + lineptr[x].r) / 2),
        static_cast<uint8_t>((lineptr_background[x].g + lineptr[x].g) / 2),
        static_cast<uint8_t>((lineptr_background[x].b + lineptr[x].b) / 2)};
}

namespace
{
    void load_logo()
    {
        static auto buffer = std::unique_ptr<std::byte, decltype(&hipFree)>{nullptr, &hipFree};

        if (buffer == nullptr)
        {
            hipError_t err;
            std::byte *ptr;
            err = hipMalloc(&ptr, logo_width * logo_height);
            CHECK_CUDA_ERROR(err);

            err = hipMemcpy(ptr, logo_data, logo_width * logo_height, hipMemcpyHostToDevice);
            CHECK_CUDA_ERROR(err);

            err = hipMemcpyToSymbol(HIP_SYMBOL(logo), &ptr, sizeof(ptr));
            CHECK_CUDA_ERROR(err);

            buffer.reset(ptr);
        }
    }
}

extern "C"
{
    void filter_impl(uint8_t *src_buffer, int width, int height, int src_stride, int pixel_stride)
    {
        load_logo();

        assert(sizeof(rgb) == pixel_stride);
        std::byte *dBuffer;
        size_t pitch;
        static int frame_count = 0;
        static std::byte *background;

        hipError_t err;

        err = hipMallocPitch(&dBuffer, &pitch, width * sizeof(rgb), height);
        CHECK_CUDA_ERROR(err);

        err = hipMemcpy2D(dBuffer, pitch, src_buffer, src_stride, width * sizeof(rgb), height, hipMemcpyDefault);
        CHECK_CUDA_ERROR(err);

        dim3 blockSize(16, 16);
        dim3 gridSize((width + (blockSize.x - 1)) / blockSize.x, (height + (blockSize.y - 1)) / blockSize.y);

        frame_count++;
        // remove_red_channel_inp<<<gridSize, blockSize>>>(dBuffer, width, height, pitch);

        if (frame_count == 1)
        {
            err = hipMallocPitch(&background, &pitch, width * sizeof(rgb), height);
            CHECK_CUDA_ERROR(err);

            err = hipMemcpy2D(background, pitch, src_buffer, src_stride, width * sizeof(rgb), height, hipMemcpyDefault);
            CHECK_CUDA_ERROR(err);
        }

        updateBackground<<<gridSize, blockSize>>>(dBuffer, background, width, height, src_stride);

        // imageDiff<<<gridSize, blockSize>>>(dBuffer, background, width, height, src_stride);

        // erosion<<<gridSize, blockSize>>>(dBuffer, width, height, src_stride);

        // end of process (dBuffer is copied into src_buffer)
        err = hipMemcpy2D(src_buffer, src_stride, dBuffer, pitch, width * sizeof(rgb), height, hipMemcpyDefault);
        CHECK_CUDA_ERROR(err);

        hipFree(dBuffer);
        hipFree(background);

        err = hipDeviceSynchronize();
        CHECK_CUDA_ERROR(err);

        {
            using namespace std::chrono_literals;
            // std::this_thread::sleep_for(100ms);
        }
    }
}
